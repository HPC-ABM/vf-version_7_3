#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Modified by Nuttiiya Seekhao to support volume rendering of float value
// from main memory

// Simple 3D volume renderer

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include "../../enums.h"


typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray[m_ecmtotal] = {0};
hipArray *d_transferFuncArrayCol = {0};
hipArray *d_transferFuncArrayEla = {0};
hipArray *d_transferFuncArrayHya = {0};

//typedef unsigned char VolumeType;
typedef float VolumeType;

texture<VolumeType, 3, hipReadModeElementType> texCol;
texture<VolumeType, 3, hipReadModeElementType> texEla;
texture<VolumeType, 3, hipReadModeElementType> texHya;
//texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<float4, 1, hipReadModeElementType>     transferTexCol; // 1D transfer function texture
texture<float4, 1, hipReadModeElementType>     transferTexEla;
texture<float4, 1, hipReadModeElementType>     transferTexHya;

typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__global__ void
d_render(uint *d_output, uint imageW, uint imageH,
         float density, float brightness,
         float transferOffset, float transferScale)
{
    const int maxSteps = 500;
    const float tstep = 0.01f;
    const float opacityThreshold = 0.95f;

    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((x >= imageW) || (y >= imageH)) return;

    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);



    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float4 sum = make_float4(0.0f);
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d*tnear;
    float3 step = eyeRay.d*tstep;


    for (int i=0; i<maxSteps; i++)
    {
        // read from 3D texture
        // remap position to [0, 1]
        float sample = tex3D(texCol, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);

        //sample *= 64.0f;    // scale for 10-bit data

        // lookup in transfer function texture
        float4 col = tex1D(transferTexCol, (sample-transferOffset)*transferScale);

        col.w *= density;

        // "under" operator for back-to-front blending
        //sum = lerp(sum, col, col.w);

        // pre-multiply alpha
        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;
        // "over" operator for front-to-back blending
        sum = sum + col*(1.0f - sum.w);

        // exit early if opaque
        if (sum.w > opacityThreshold)
            break;

        t += tstep;

        if (t > tfar) break;

        pos += step;

    }

    sum *= brightness;

    // write output color
    d_output[y*imageW + x] = rgbaFloatToInt(sum);
}

__global__ void
d_render_dim(uint *d_output, uint nx, uint ny, uint nz, uint imageW, uint imageH,
         float density, float brightness,
         float transferOffset, float transferScale, ecm_i ecmType)
{
    const int maxSteps = 500;
    const float tstep = 0.01f;
    const float opacityThreshold = 0.95f;

    // Calculate box dimensions using largest dimension as reference
    const float a = -1.0f;
    const float b = +1.0f;
    const float ref = (float) max(nx, max(ny, nz));
    const float x_halfwidth = (((float) nx)/(2.0f * ref))*(b-a);
    const float y_halfwidth = (((float) ny)/(2.0f * ref))*(b-a);
    const float z_halfwidth = (((float) nz)/(2.0f * ref))*(b-a);

    const float3 boxMin = make_float3(-1.0f*x_halfwidth, -1.0f*y_halfwidth, -1.0f*z_halfwidth);
    const float3 boxMax = make_float3( 1.0f*x_halfwidth,  1.0f*y_halfwidth,  1.0f*z_halfwidth);

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((x >= imageW) || (y >= imageH)) return;

    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);


    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float4 sum = make_float4(0.0f);
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d*tnear;
    float3 step = eyeRay.d*tstep;


    for (int i=0; i<maxSteps; i++)
    {
        // read from 3D texture
        // remap position to [0, 1]

    	  float posx = (pos.x + x_halfwidth)/(2.0f*x_halfwidth);
    	  float posy = (pos.y + y_halfwidth)/(2.0f*y_halfwidth);
    	  float posz = (pos.z + z_halfwidth)/(2.0f*z_halfwidth);

    	  float sample;
    	  float4 col;

    	  switch (ecmType)
    	  {
    	  case m_col:
    	  	sample = tex3D(texCol, posx, posy, posz);

    	  	//sample *= 64.0f;    // scale for 10-bit data

    	  	// lookup in transfer function texture
    	  	col = tex1D(transferTexCol, (sample-transferOffset)*transferScale);
    	  	break;
    	  case m_ela:
    	  	sample = tex3D(texEla, posx, posy, posz);
    	  	// lookup in transfer function texture
    	  	col = tex1D(transferTexEla, (sample-transferOffset)*transferScale);
    	  	break;
    	  case m_hya:
    	  	sample = tex3D(texHya, posx, posy, posz);
    	  	// lookup in transfer function texture
    	  	col = tex1D(transferTexHya, (sample-transferOffset)*transferScale);
    	  	break;
    	  }

        col.w *= density;

        // "under" operator for back-to-front blending
        //sum = lerp(sum, col, col.w);

        // pre-multiply alpha
        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;
        // "over" operator for front-to-back blending
        sum = sum + col*(1.0f - sum.w);

        // exit early if opaque
        if (sum.w > opacityThreshold)
            break;

        t += tstep;

        if (t > tfar) break;

        pos += step;

    }

    sum *= brightness;

    // write output color
    d_output[y*imageW + x] = rgbaFloatToInt(sum);
}




extern "C"
void setTextureFilterMode(bool bLinearFilter)
{
    texCol.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
    texEla.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
    texHya.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}

extern "C"
void bufferECMmap(hipMemcpy3DParms copyParams)
{
	checkCudaErrors(hipMemcpy3D(&copyParams));
}

extern "C"
void initCuda(void *h_volume, hipExtent volumeSize, hipMemcpy3DParms &copyParams, ecm_i ecmType)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors(hipMalloc3DArray(&(d_volumeArray[ecmType]), &channelDesc, volumeSize));

    // copy data to 3D array
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray[ecmType];
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    //create transfer function texture
    switch(ecmType)
		{
    case m_col:
    {
    	// Collagen
    	float4 transferFunc[] =
    	{
    			{  0.00,  0.00,  0.00, 0.0, },	// 0.00
    			{  1.00,  0.00,  0.00, 0.5, },	// 0.05 - SLP ILP
    			{  1.00,  0.30,  0.30, 0.8, },	// 0.10 - SLP ILP
    			{  0.00,  0.00,  0.00, 0.0, },	// 0.15 - SLP ILP
    			{  0.80,  0.15,  0.10, 0.2, }, // 0.20 - DLP ILP SLP
    			{  0.80,  0.15,  0.10, 0.3, }, // 0.25 - DLP
    			{  0.80,  0.15,  0.10, 0.4, }, // 0.30 - DLP
    			{  0.80,  0.15,  0.10, 0.5, }, // 0.35 - DLP
    			{  0.80,  0.15,  0.10, 0.6, }, // 0.40 - DLP
    			{  0.80,  0.15,  0.10, 0.7, }, // 0.45
    			{  0.80,  0.15,  0.10, 0.8, }, // 0.50
    			{  0.80,  0.15,  0.10, 0.9, }, // 0.55
    			{  0.85,  0.10,  0.15, 0.8, }, // 0.60
    			{  0.90,  0.10,  0.15, 0.7, }, // 0.65
    			{  0.95,  0.10,  0.10, 0.6, }, // 0.70
    			{  1.00,  0.10,  0.10, 0.5, }, // 0.75
    			{  1.00,  0.10,  0.10, 0.6, }, // 0.80
    			{  1.00,  0.10,  0.10, 0.7, }, // 0.85
    			{  1.00,  0.20,  0.20, 0.8, }, // 0.90
    			{  1.00,  0.30,  0.30, 0.9, }, // 0.95
    			{  1.00,  0.60,  0.00, 1.0, }, // 1.00
    			{  0.60,  0.40,  0.32, 1.0, },
    	};
//    	float4 transferFunc[] =
//    	{
//    			{  0.0,  0.0,  0.0, 0.0, },	// 0.00
//    			{  1.0,  0.0,  0.0, 0.5, },	// 0.05 - SLP ILP
//    			{  0.0,  0.0,  0.0, 0.0, },	// 0.10 - SLP ILP
//    			{  0.0,  0.0,  0.0, 0.0, },	// 0.15 - SLP ILP
//    			{  0.0,  0.0,  0.0, 0.0, }, // 0.20 - DLP ILP SLP
//    			{  0.0,  0.0,  0.0, 0.0, }, // 0.25 - DLP
//    			{  1.0,  0.0,  0.0, 0.2, }, // 0.30 - DLP
//    			{  1.0,  0.0,  0.0, 0.2, }, // 0.35 - DLP
//    			{  1.0,  0.0,  0.0, 0.2, }, // 0.40 - DLP
//    			{  1.0,  0.0,  0.0, 0.2, }, // 0.45
//    			{  1.0,  0.0,  0.0, 0.2, }, // 0.50
//    			{  1.0,  0.0,  0.0, 0.3, }, // 0.55
//    			{  1.0,  0.0,  0.0, 0.4, }, // 0.60
//    			{  1.0,  0.0,  0.0, 0.5, }, // 0.65
//    			{  1.0,  0.0,  0.0, 0.6, }, // 0.70
//    			{  1.0,  0.0,  0.0, 0.7, }, // 0.75
//    			{  1.0,  0.0,  0.0, 0.8, }, // 0.80
//    			{  1.0,  0.1,  0.1, 1.0, }, // 0.85
//    			{  1.0,  0.2,  0.2, 0.5, }, // 0.90
//    			{  1.0,  0.3,  0.3, 0.7, }, // 0.95
//    			{  1.0,  0.6,  0.0, 0.8, }, // 1.00
//    	};
      // set texture parameters
      texCol.normalized = true;                      // access with normalized texture coordinates
      texCol.filterMode = hipFilterModeLinear;      // linear interpolation
      texCol.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
      texCol.addressMode[1] = hipAddressModeClamp;

      // bind array to 3D texture
      checkCudaErrors(hipBindTextureToArray(texCol, d_volumeArray[ecmType], channelDesc));

      hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
      hipArray *d_transferFuncArrayCol;
      checkCudaErrors(hipMallocArray(&d_transferFuncArrayCol, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
      checkCudaErrors(hipMemcpyToArray(d_transferFuncArrayCol, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

      transferTexCol.filterMode = hipFilterModeLinear;
      transferTexCol.normalized = true;    // access with normalized texture coordinates
      transferTexCol.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

      // Bind the array to the texture
      checkCudaErrors(hipBindTextureToArray(transferTexCol, d_transferFuncArrayCol, channelDesc2));
    	break;
    }

    case m_ela:
    {
      // Elastin
      float4 transferFunc[] =
      {
      		{  0.0,  0.0,  0.0, 0.0, },	// 0.00
      		{  0.0,  1.0,  0.0, 0.5, },	// 0.05 - SLP ILP
      		{  0.0,  0.0,  0.0, 0.0, },	// 0.10 - SLP ILP
      		{  0.63, 0.12,  0.4, 0.3, },	// 0.15 - SLP ILP
      		{  0.63, 0.12,  0.4, 0.4, }, // 0.20 - DLP ILP SLP
      		{  0.63, 0.12,  0.4, 0.5, }, // 0.25 - DLP
      		{  0.63, 0.12,  0.4, 0.6, }, // 0.30 - DLP
      		{  0.63, 0.12,  0.4, 0.7, }, // 0.35 - DLP
      		{  0.63, 0.12,  0.4, 0.8, }, // 0.40 - DLP
      		{  0.63, 0.12,  0.4, 0.9, }, // 0.45
      		{  0.0, 1.0,  0.30, 1.0, }, // 0.50
      		{  0.0, 1.0,  0.30, 1.0, }, // 0.55
      		{  0.0, 1.0,  0.30, 1.0, }, // 0.60
      		{  0.0, 1.0,  0.30, 1.0, }, // 0.65
      		{  0.0, 1.0,  0.30, 1.0, }, // 0.70
      		{  0.0, 1.0,  0.30, 1.0, }, // 0.75
      		{  0.0,  0.0,  0.0, 0.0, }, // 0.80
      		{  0.0, 1.0,  0.30, 1.0, }, // 0.85
      		{  0.0, 1.0,  0.40, 0.5, }, // 0.90
      		{  0.0, 1.0,  0.50, 0.7, }, // 0.95
      		{  0.0, 1.0,  0.60, 1.0, }, // 1.00
      };
      // set texture parameters
      texEla.normalized = true;                      // access with normalized texture coordinates
      texEla.filterMode = hipFilterModeLinear;      // linear interpolation
      texEla.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
      texEla.addressMode[1] = hipAddressModeClamp;

      // bind array to 3D texture
      checkCudaErrors(hipBindTextureToArray(texEla, d_volumeArray[ecmType], channelDesc));

      hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
      hipArray *d_transferFuncArrayEla;
      checkCudaErrors(hipMallocArray(&d_transferFuncArrayEla, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
      checkCudaErrors(hipMemcpyToArray(d_transferFuncArrayEla, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

      transferTexEla.filterMode = hipFilterModeLinear;
      transferTexEla.normalized = true;    // access with normalized texture coordinates
      transferTexEla.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

      // Bind the array to the texture
      checkCudaErrors(hipBindTextureToArray(transferTexEla, d_transferFuncArrayEla, channelDesc2));

    	break;
    }

    case m_hya:
    {
      // Hyaluronan
      float4 transferFunc[] =
      {
      		{  0.0,  0.00,  0.00, 0.0, },	// 0.00
      		{  0.0,  0.00,  1.00, 0.5, },	// 0.05 - SLP ILP
      		{  0.3,  0.30,  1.00, 0.8, },	// 0.10 - SLP ILP
      		{  0.0,  0.00,  0.00, 0.0, },	// 0.15 - SLP ILP
      		{  1.0,  0.43,  0.78, 0.2, }, // 0.20 - DLP ILP SLP
      		{  1.0,  0.43,  0.78, 0.3, }, // 0.25 - DLP
      		{  1.0,  0.43,  0.78, 0.4, }, // 0.30 - DLP
      		{  1.0,  0.43,  0.78, 0.5, }, // 0.35 - DLP
      		{  1.0,  0.43,  0.78, 0.6, }, // 0.40 - DLP
      		{  1.0,  0.43,  0.78, 0.7, }, // 0.45
      		{  1.0,  0.43,  0.78, 0.8, }, // 0.50
      		{  1.0,  0.43,  0.78, 0.9, }, // 0.55
      		{  0.8,  0.33,  0.85, 0.8, }, // 0.60
      		{  0.5,  0.23,  0.90, 0.7, }, // 0.65
      		{  0.3,  0.13,  0.95, 0.6, }, // 0.70
      		{  0.0,  0.00,  1.00, 0.5, }, // 0.75
      		{  0.1,  0.10,  1.00, 0.6, }, // 0.80
      		{  0.2,  0.20,  1.00, 0.7, }, // 0.85
      		{  0.3,  0.30,  1.00, 0.8, }, // 0.90
      		{  0.4,  0.40,  1.00, 0.9, }, // 0.95
      		{  0.7,  0.70,  1.00, 1.0, }, // 1.00
      };
      // set texture parameters
      texHya.normalized = true;                      // access with normalized texture coordinates
      texHya.filterMode = hipFilterModeLinear;      // linear interpolation
      texHya.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
      texHya.addressMode[1] = hipAddressModeClamp;

      // bind array to 3D texture
      checkCudaErrors(hipBindTextureToArray(texHya, d_volumeArray[ecmType], channelDesc));

      hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
      hipArray *d_transferFuncArrayHya;
      checkCudaErrors(hipMallocArray(&d_transferFuncArrayHya, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
      checkCudaErrors(hipMemcpyToArray(d_transferFuncArrayHya, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

      transferTexHya.filterMode = hipFilterModeLinear;
      transferTexHya.normalized = true;    // access with normalized texture coordinates
      transferTexHya.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

      // Bind the array to the texture
      checkCudaErrors(hipBindTextureToArray(transferTexHya, d_transferFuncArrayHya, channelDesc2));


    	break;
    }

    default:
    {
    	// WEF
    	// White bg
    	float4 transferFunc[] =
    	{
    			{  0.0,  0.0,  0.0, 0.0, },
    			{  1.0,  0.0,  0.0, 1.0, },
    			{  1.0,  0.0,  0.0, 1.0, },
    			{  0.97, 0.8, 0.72, 1.0, },
    			{  0.97, 0.8, 0.72, 0.5, },
    			{  0.80, 0.6, 0.52, 0.7, },
    			{  0.60, 0.4, 0.32, 1.0, },//0.5, },
    	};

//    	// Black bg
//    	float4 transferFunc[] =
//    	{
//    			{  0.0,  0.0,  0.0, 0.0, },
//    			{  1.0,  0.0,  0.0, 1.0, },
//    			{  1.0,  0.0,  0.0, 1.0, },
//    			{  0.97, 0.8, 0.72, 1.0, },
//    			{  0.97, 0.4, 0.30, 1.0, },
//    			{  0.97, 0.6, 0.50, 0.7, },
//    			{  0.97, 0.8, 0.72, 0.8, },//0.5, },
//		  };
      // set texture parameters
      texCol.normalized = true;                      // access with normalized texture coordinates
      texCol.filterMode = hipFilterModeLinear;      // linear interpolation
      texCol.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
      texCol.addressMode[1] = hipAddressModeClamp;

      // bind array to 3D texture
      checkCudaErrors(hipBindTextureToArray(texCol, d_volumeArray[ecmType], channelDesc));

      hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
      hipArray *d_transferFuncArrayCol;
      checkCudaErrors(hipMallocArray(&d_transferFuncArrayCol, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
      checkCudaErrors(hipMemcpyToArray(d_transferFuncArrayCol, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

      transferTexCol.filterMode = hipFilterModeLinear;
      transferTexCol.normalized = true;    // access with normalized texture coordinates
      transferTexCol.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

      // Bind the array to the texture
      checkCudaErrors(hipBindTextureToArray(transferTexCol, d_transferFuncArrayCol, channelDesc2));

    	break;
    }

		}


//        float4 transferFunc[] =
//        {
//        		{  0.0, 0.0, 0.0, 0.0, },
//        		{  1.0, 0.0, 0.0, 0.1, },
//        		{  0.0, 0.0, 0.0, 0.0, },
//        		{  0.0, 0.0, 0.0, 0.0, },
//        		{  1.0, 0.0, 0.0, 0.00001, },//0.1
//        		{  0.97, 0.8, 0.72, 0.01, },
//        		{  0.0, 0.0, 0.0, 0.001, },
//        		{  0.0, 0.0, 0.0, 0.001, },
//        		{  0.0, 0.0, 0.0, 0.001, },		// red
//        		{  1.0, 0.0, 0.0, 0.001, },		// red
//        		{  0.97, 0.8, 0.72, 0.01, },		// red
//        		{  0.97, 0.8, 0.72, 0.01, },		// red
//        		//    		{  1.0, 0.0, 0.0, 0.1, },		// red
//        		{  0.97, 0.8, 0.72, 0.12, },		// flesh
//        };


//    float4 transferFunc[] =
//    {
//    		{  0.0, 0.0, 0.0, 0.0, },
//    		{  1.0, 0.0, 0.0, 0.1, },
//    		{  0.0, 0.0, 0.0, 0.0, },
//    		{  0.0, 0.0, 0.0, 0.0, },
//    		{  1.0, 0.0, 0.0, 0.00001, },//0.1
//    		{  0.97, 0.8, 0.72, 0.01, },
//    		{  0.0, 0.0, 0.0, 0.001, },
//    		{  0.0, 0.0, 0.0, 0.001, },
//    		{  0.0, 0.0, 0.0, 0.001, },		// red
//    		{  1.0, 0.0, 0.0, 0.001, },		// red
//    		{  0.97, 0.8, 0.72, 0.01, },		// red
//    		{  0.97, 0.8, 0.72, 0.01, },		// red
//    		//    		{  1.0, 0.0, 0.0, 0.1, },		// red
//    		{  0.97, 0.8, 0.72, 0.12, },		// flesh
//    };
//    float4 transferFunc[] =
//    {
//        {  0.5, 0.0, 0.2, 0.0 },      // 0.0
//        {  0.5, 0.0, 0.2, 0.100 },
//        {0.576, 1.439, 1.000, 0.1000 },  // 0.1
//        {1.000, 0.753, 0.796, 0.300 }, // 0.2
//        {0.000, 0.749, 1.000, 0.400 },  // 0.3
//        {0.000, 0.749, 1.000, 0.500 },  // 0.4
//        {0.498, 1.000, 0.831, 0.600 },  // 0.5
//        {1.000, 0.489, 0.314, 0.700 }, // 0.6
//        {1.000, 0.750, 0.700, 0.750 }, // 0.7
//        {1.000, 1.000, 0.000, 0.800 }, // 0.8
//        {0.500, 1.000, 0.200, 0.900 }, // 0.9
//        {0.000, 1.000, 0.000, 1.000 }, // 1.0
//        {0.999, 1.000, 0.999, 1.000 }, // 1.0
//    };



}

extern "C"
void freeCudaBuffers()
{
	for(int ei = 0; ei < m_ecmtotal; ei++) {
    checkCudaErrors(hipFreeArray(d_volumeArray[ei]));
	}

	checkCudaErrors(hipFreeArray(d_transferFuncArrayCol));
	checkCudaErrors(hipFreeArray(d_transferFuncArrayEla));
	checkCudaErrors(hipFreeArray(d_transferFuncArrayHya));
}

extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH,
                   float density, float brightness, float transferOffset, float transferScale)
{
    d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, density,
                                      brightness, transferOffset, transferScale);
}

extern "C"
void render_kernel_dim(dim3 gridSize, dim3 blockSize, uint *d_output, uint nx, uint ny, uint nz, uint imageW, uint imageH,
                   float density, float brightness, float transferOffset, float transferScale, ecm_i ecmType)
{
    d_render_dim<<<gridSize, blockSize>>>(d_output, nx, ny, nz, imageW, imageH, density,
                                      brightness, transferOffset, transferScale, ecmType);
}

extern "C"
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}


#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
